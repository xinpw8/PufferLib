#include "hip/hip_runtime.h"
#include <torch/extension.h>

__global__ void advantage_kernel(
    float* reward_block,    // [num_steps, horizon]
    float* reward_mask,     // [num_steps, horizon]
    float* values_mean,     // [num_steps, horizon]
    float* values_std,      // [num_steps, horizon]
    float* buf,            // [num_steps, horizon]
    float* dones,          // [num_steps]
    float* rewards,        // [num_steps]
    float* advantages,     // [num_steps]
    int* bounds,          // [num_steps]
    int num_steps,
    int horizon,
    float vstd_min,
    float vstd_max
) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= num_steps) return;

    int k = 0;
    float adv_sum = 0.0f;
    float delta = vstd_min - vstd_max;

    for (int j = 0; j < horizon; j++) {
        int t = i + j;
        if (t >= num_steps - 1 || dones[t]) {
            break;
        }
        k = j + 1;

        int idx = i * horizon + j;
        reward_block[idx] = rewards[t + 1];
        reward_mask[idx] = 1.0f;

        float vstd = values_std[idx];
        buf[idx] = vstd;

        float adv_scale = (delta == 0) ? 1.0f : (vstd_max - vstd) / delta;
        adv_scale = max(min(adv_scale, 1.0f), 0.05f);
        buf[idx] = adv_scale;
        adv_sum += adv_scale;
    }

    bounds[i] = k;

    if (delta == 0) {
        float adv = 0.0f;
        for (int j = 0; j < k; j++) {
            int idx = i * horizon + j;
            adv += (reward_block[idx] - values_mean[idx]);
        }
        advantages[i] = adv;
        return;
    }

    float adv = 0.0f;
    for (int j = 0; j < k; j++) {
        int idx = i * horizon + j;
        adv += (buf[idx] / adv_sum) * (reward_block[idx] - values_mean[idx]);
    }
    advantages[i] = adv;
}

// Pybind11 module definition
PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("advantage_kernel", [](torch::Tensor reward_block,
                                torch::Tensor reward_mask,
                                torch::Tensor values_mean,
                                torch::Tensor values_std,
                                torch::Tensor buf,
                                torch::Tensor dones,
                                torch::Tensor rewards,
                                torch::Tensor advantages,
                                torch::Tensor bounds,
                                int num_steps,
                                int horizon,
                                float vstd_max,
                                float vstd_min) {
        // Launch the kernel
        int threads_per_block = 256;
        int blocks = (num_steps + threads_per_block - 1) / threads_per_block;

        advantage_kernel<<<blocks, threads_per_block>>>(
            reward_block.data_ptr<float>(),
            reward_mask.data_ptr<float>(),
            values_mean.data_ptr<float>(),
            values_std.data_ptr<float>(),
            buf.data_ptr<float>(),
            dones.data_ptr<float>(),
            rewards.data_ptr<float>(),
            advantages.data_ptr<float>(),
            bounds.data_ptr<int>(),
            num_steps,
            horizon,
            vstd_max,
            vstd_min
        );

        // Check for CUDA errors
        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            throw std::runtime_error(hipGetErrorString(err));
        }
    }, "Compute advantages with CUDA");
}
